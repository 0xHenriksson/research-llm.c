/*
Kernels for softmax forward pass.

Compile example:
nvcc -O3 --use_fast_math softmax_forward.cu -o softmax_forward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./softmax_forward 1

version 2 is a fused kernel that parallelizes over all of B,T,C
./softmax_forward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------
// CUDA utils

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// error checking
void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// ----------------------------------------------------------------------------
// CPU code reference

void softmax_forward_cpu(float* out, float* inp, int N, int C) {
    // inp is (N, C)
    // out is (N, C), each row of inp will get softmaxed
    for (int i = 0; i < N; i++) {
        float* inp_row = inp + i * C;
        float* out_row = out + i * C;

        float maxval = -INFINITY;
        for (int j = 0; j < C; j++) {
            if (inp_row[j] > maxval) {
                maxval = inp_row[j];
            }
        }
        float sum = 0.0f;
        for (int j = 0; j < C; j++) {
            out_row[j] = expf(inp_row[j] - maxval);
            sum += out_row[j];
        }
        for (int j = 0; j < C; j++) {
            out_row[j] /= sum;
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void softmax_forward_kernel1(float* out, float* inp, int N, int C) {
    // inp is (N, C)
    // out is (N, C), each row of inp will get softmaxed
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float* inp_row = inp + i * C;
        float* out_row = out + i * C;

        float maxval = -INFINITY;
        for (int j = 0; j < C; j++) {
            if (inp_row[j] > maxval) {
                maxval = inp_row[j];
            }
        }
        float sum = 0.0f;
        for (int j = 0; j < C; j++) {
            out_row[j] = expf(inp_row[j] - maxval);
            sum += out_row[j];
        }
        for (int j = 0; j < C; j++) {
            out_row[j] /= sum;
        }
    }
}

__global__ void softmax_forward_kernel2(float* out, float* inp, int N, int C) {
    // inp is (N, C)
    // in each row of C elements, first calculates maxval, then returns expf(val - maxval)
    extern __shared__ float shared[];
    int idx = blockIdx.x; // ranges [0, N)
    int tid = threadIdx.x; // ranges [0, block_size)
    int block_size = blockDim.x;
    float* x = inp + idx * C; // idx-th row of inp
    // thread coarsening
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += block_size) {
        maxval = fmaxf(maxval, x[i]);
    }
    shared[tid] = maxval;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] = fmaxf(shared[tid], shared[tid + stride]);
        }
    }
    float offset = shared[0];
    __syncthreads();
    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += block_size) {
        out[idx * C + i] = expf(x[i] - offset);
    }
    __syncthreads();
    // thread coarsening again, for the sum
    x = out + idx * C; // idx-th row of out
    float sumval = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        sumval += x[i];
    }
    shared[tid] = sumval;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // broadcast the sum to all threads in the block
    __syncthreads();
    float sum = shared[0];
    // divide the input values by the sum
    for (int i = tid; i < C; i += block_size) {
        out[idx * C + i] = x[i] / sum;
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void softmax_forward1(float* out, float* inp, int N, int C, const int block_size) {
    const int grid_size = CEIL_DIV(N, block_size);
    softmax_forward_kernel1<<<grid_size, block_size>>>(out, inp, N, C);
    cudaCheck(hipGetLastError());
}

void softmax_forward2(float* out, float* inp, int N, int C, const int block_size) {
    int grid_size = N;
    size_t shared_mem_size = block_size * sizeof(float);
    softmax_forward_kernel2<<<grid_size, block_size, shared_mem_size>>>(out, inp, N, C);
}

// kernel version dispatch
void softmax_forward(int kernel_num, float* out, float* inp, int N, int C, const int block_size) {
    switch (kernel_num) {
        case 1:
            softmax_forward1(out, inp, N, C, block_size);
            break;
        case 2:
            softmax_forward2(out, inp, N, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------
// random utils

float* make_random_float(int N) {
    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        arr[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
    }
    return arr;
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * T * sizeof(float));
    float* inp = make_random_float(B * T * T);

    // move to GPU
    float* d_out;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * T * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * T * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    float* out_gpu = (float*)malloc(B * T * T * sizeof(float));

    // first check the correctness of the kernel
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        softmax_forward_cpu(out, inp, B * T, T);
        softmax_forward(kernel_num, d_out, d_inp, B * T, T, block_size);
        cudaCheck(hipMemcpy(out_gpu, d_out, B * T * T * sizeof(float), hipMemcpyDeviceToHost));
        for (int i = 0; i < B * T * T; i++) {
            // print the first few comparisons
            if (i < 5) {
                printf("%f %f\n", out[i], out_gpu[i]);
            }
            // ensure correctness for all elements
            if (fabs(out[i] - out_gpu[i]) > 1e-4) {
                printf("Mismatch at %d: %f vs %f\n", i, out[i], out_gpu[i]);
                exit(1);
            }
        }
        printf("Results match at block_size=%d\n", block_size);
    }

    // time the kernel at different block sizes
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];

        int repeat_times = 1000;
        hipEvent_t start, stop;
        cudaCheck(hipEventCreate(&start));
        cudaCheck(hipEventCreate(&stop));
        cudaCheck(hipEventRecord(start, 0));
        for (int i = 0; i < repeat_times; i++) {
            softmax_forward(kernel_num, d_out, d_inp, B * T, T, block_size);
        }
        cudaCheck(hipEventRecord(stop, 0));
        cudaCheck(hipEventSynchronize(start));
        cudaCheck(hipEventSynchronize(stop));
        float elapsed_time;
        cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time / repeat_times);
    }

    // free memory
    free(out);
    free(inp);
    free(out_gpu);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));

    return 0;
}